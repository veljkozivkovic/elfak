
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 8

typedef struct Tacka {
    float x;
    float y;
    int index;
} Tacka;

__device__ float distance(Tacka t1, Tacka t2) {
    return sqrtf(powf(t1.x - t2.x, 2) + powf(t1.y - t2.y, 2));
}

__global__ void min_reduction(Tacka* a, Tacka* b, Tacka* ref_tacka) {
    __shared__ Tacka tacke[BLOCK_SIZE];
    Tacka ref = *ref_tacka;
    int i = threadIdx.x + blockIdx.x * blockDim.x * 2;
    if (distance(a[i], ref) < distance(a[i + blockDim.x], ref))
        tacke[threadIdx.x] = a[i];
    else
        tacke[threadIdx.x] = a[i + blockDim.x];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s && 
            distance(tacke[threadIdx.x + s], ref) < distance(tacke[threadIdx.x], ref))
            tacke[threadIdx.x] = tacke[threadIdx.x + s];
        __syncthreads();
    }

    if (threadIdx.x == 0)
        b[blockIdx.x] = tacke[0];
}

__host__ void get_10_min(Tacka* tacke, int n) {
    int grid_size = n / BLOCK_SIZE / 2;
    Tacka* dev_a, *dev_b, *ref_tacka;
    hipMalloc((void**)&dev_a, n * sizeof(Tacka));
    hipMalloc((void**)&dev_b, n * sizeof(Tacka));
    hipMalloc((void**)&ref_tacka, sizeof(Tacka));
    hipMemcpy(ref_tacka, tacke, sizeof(Tacka), hipMemcpyHostToDevice);
    Tacka reduced_tacka;

    for (int i = 1; i < 11; ++i) {
        hipMemcpy(dev_a, &tacke[i], n * sizeof(Tacka), hipMemcpyHostToDevice);

        min_reduction<<<grid_size, BLOCK_SIZE>>>(dev_a, dev_b, ref_tacka);
        min_reduction<<<1, grid_size / 2>>>(dev_b, dev_b, ref_tacka);

        hipMemcpy(&reduced_tacka, dev_b, sizeof(Tacka), hipMemcpyDeviceToHost);

        tacke[reduced_tacka.index] = tacke[i];
        tacke[reduced_tacka.index].index = reduced_tacka.index;
        tacke[i] = reduced_tacka;
        tacke[i].index = i;
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(ref_tacka);
}

int get_arr_len(int n) {
    if (n <= 4 * BLOCK_SIZE)
        return 4 * BLOCK_SIZE;
    int p = 1;
    while (p < n)
        p <<= 1;
    return p;
}

int main() {
    int n = 25;
    int N = get_arr_len(n);

    Tacka* tacke = (Tacka*)malloc((N + 12) * sizeof(Tacka));
    for (int i = 0; i < n; ++i) {
        tacke[i].x = i;
        tacke[i].y = i;
        tacke[i].index = i;
    }
    for (int i = n; i < N + 12; ++i) {
        tacke[i].x = sqrtf(INT_MAX) - 1;
        tacke[i].y = sqrtf(INT_MAX) - 1;
        tacke[i].index = i;
    }
    get_10_min(tacke, N);
    for (int i = 1; i < 11; ++i) {
        printf("%5.2f %5.2f\n", tacke[i].x, tacke[i].y);
    }

    free(tacke);
}