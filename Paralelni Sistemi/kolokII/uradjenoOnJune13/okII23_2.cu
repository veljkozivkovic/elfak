
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256
#define GRID_SIZE 256
int elements_per_reduction = BLOCK_SIZE * GRID_SIZE * 2;

__global__ void get_c(int* a, int* b, int* c, int* n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int N = *n;
    while (tid < N * N) {
        int row = tid / N;
        int col = tid % N;
        int b_ind = col * N + row;
        c[tid] = min(a[tid], b[b_ind]);
        tid += blockDim.x * gridDim.x;
    }
}

__host__ void get_matrix_c(int* a, int* b, int* c, int n) {
    int* dev_a, *dev_b, *dev_c, *dev_n;
    hipMalloc((void**)&dev_a, n * n * sizeof(int));
    hipMalloc((void**)&dev_b, (n + 1) * (n + 1) * sizeof(int));
    hipMalloc((void**)&dev_c, n * n * sizeof(int));
    hipMalloc((void**)&dev_n, sizeof(int));

    hipMemcpy(dev_a, a, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);

    get_c<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b, dev_c, dev_n);

    hipMemcpy(c, dev_c, n * n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_c);
    hipFree(dev_b);
    hipFree(dev_n);
}

__global__ void sum_reduction(int* a, int* b) {
    __shared__ int partial_sum[BLOCK_SIZE];
    int i = threadIdx.x + blockIdx.x * blockDim.x * 2;
    partial_sum[threadIdx.x] = a[i] + a[i + blockDim.x];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s)
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        __syncthreads();
    }

    if (threadIdx.x == 0)
        b[blockIdx.x] = partial_sum[0];
}

__host__ int get_sum(int* a, int N) {
    int* dev_a, *dev_b;
    hipMalloc((void**)&dev_a, elements_per_reduction * sizeof(int));
    hipMalloc((void**)&dev_b, BLOCK_SIZE * sizeof(int));

    int s = 0;
    int reduced_sum;
    for (int i = 0; i < N; i += elements_per_reduction) {
        hipMemcpy(dev_a, &a[i], elements_per_reduction * sizeof(int), hipMemcpyHostToDevice);

        sum_reduction<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b);
        sum_reduction<<<1, GRID_SIZE / 2>>>(dev_b, dev_b);

        hipMemcpy(&reduced_sum, dev_b, sizeof(int), hipMemcpyDeviceToHost);
        s += reduced_sum;
    }

    hipFree(dev_a);
    hipFree(dev_b);

    return s;
}

int get_arr_len(int n) {
    if (n <= elements_per_reduction)
        return elements_per_reduction;
    int p = 1;
    while (p < n)
        p <<= 1;
    return p;
}

int main() {
    int n = 5;
    int* a = (int*)malloc(n * n * sizeof(int));
    int* b = (int*)malloc(n * n * sizeof(int));
    int* c = (int*)malloc(n * n * sizeof(int));

    for (int i = 0; i < n * n; ++i) {
        a[i] = 10 + rand() % 89;
        b[i] = 10 + rand() % 89;
    }

    get_matrix_c(a, b, c, n);

    int N = get_arr_len(n);
    float* avgs = (float*)malloc(n * sizeof(float));
    int* col = (int*)malloc(N * sizeof(int));

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            col[j] = c[j * n + i];
        }
        for (int j = n; j < N; ++j) {
            col[j] = 0;
        }
        int s = get_sum(col, N);
        avgs[i] = (float)s / n;
    }

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%5d ", c[i * n + j]);
        }
        printf("\n");
    }
    for (int i = 0; i < n; ++i) {
        printf("%5.2f ", avgs[i]);
    }

    free(a);
    free(b);
    free(c);
    free(avgs);
    free(col);

    return 0;
}