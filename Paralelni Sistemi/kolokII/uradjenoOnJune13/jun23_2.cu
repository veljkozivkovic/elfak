#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256
#define GRID_SIZE 256
int elements_per_reduction = BLOCK_SIZE * GRID_SIZE * 2;

__global__ void min_reduction(int* a, int* b) {
    __shared__ int partial_min[BLOCK_SIZE];
    int i = threadIdx.x + blockIdx.x * blockDim.x * 2;
    partial_min[threadIdx.x] = min(a[i], a[i + blockDim.x]);
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s)
            partial_min[threadIdx.x] = min(partial_min[threadIdx.x], partial_min[threadIdx.x + s]);
        __syncthreads();
    }

    if (threadIdx.x == 0)
        b[blockIdx.x] = partial_min[0];
}

__host__ int get_min(int* a, int N) {
    int* dev_a, *dev_b;
    hipMalloc((void**)&dev_a, elements_per_reduction * sizeof(int));
    hipMalloc((void**)&dev_b, BLOCK_SIZE * sizeof(int));

    int reduced_min;
    int min_ = INT_MAX;
    for (int i = 0; i < N; i += elements_per_reduction) {
        hipMemcpy(dev_a, &a[i], elements_per_reduction * sizeof(int), hipMemcpyHostToDevice);

        min_reduction<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b);
        min_reduction<<<1, GRID_SIZE / 2>>>(dev_b, dev_b);

        hipMemcpy(&reduced_min, dev_b, sizeof(int), hipMemcpyDeviceToHost);
        min_ = min(reduced_min, min_);
    }

    hipFree(dev_a);
    hipFree(dev_b);

    return min_;
}

int get_arr_len(int n) {
    if (n <= elements_per_reduction)
        return elements_per_reduction;
    int p = 1;
    while (p < n) {
        p <<= 1;
    }
    return p;
}

int main() {
    int n = 25;

    int* matrix = (int*)malloc(n * n * sizeof(int));
    int actual_min = INT_MAX;

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            matrix[i * n + j] = rand();
            if (i == j) actual_min = min(actual_min, matrix[i * n + j]);
        }
    }

    int N = get_arr_len(n);
    int* diag = (int*)malloc(N * sizeof(int));
    for (int i = 0; i < n; ++i)
        diag[i] = matrix[i * n + i];
    for (int i = n; i < N; ++i)
        diag[i] = INT_MAX;
    
    int reduced_min = get_min(diag, N);
    printf("%d == %d\n", actual_min, reduced_min);

    free(matrix);
    free(diag);
}