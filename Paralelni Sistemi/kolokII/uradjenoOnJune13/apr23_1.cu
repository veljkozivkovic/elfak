
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_DIM 4

__global__ void anti_aliasing_gpu(float* a, int* N) {
    __shared__ float shm[BLOCK_DIM + 2][BLOCK_DIM + 2];
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int n = *N;
    int dirs[3] = { 1, 0, -1 };

    if (x >= n || y >= n)
        return;
    
    int tid = y * n + x;
    shm[threadIdx.y + 1][threadIdx.x + 1] = a[tid];
    if (threadIdx.y == 0 && y > 0) {
        int new_element = tid - n;
        shm[threadIdx.y][threadIdx.x + 1] = a[new_element];
        if (threadIdx.x == 0 && x > 0) {
            new_element -= 1;
            shm[threadIdx.y][threadIdx.x] = a[new_element];
        }
        else if (threadIdx.x == blockDim.x && x < gridDim.x - 1) {
            new_element += 1;
            shm[threadIdx.y][threadIdx.x + 2] = a[new_element];
        }
    }
    if (threadIdx.y == blockDim.y && y < gridDim.y - 1) {
        int new_element = tid + n;
        shm[threadIdx.y + 2][threadIdx.x + 1] = a[new_element];
        if (threadIdx.x == 0 && x > 0) {
            new_element -= 1;
            shm[threadIdx.y + 2][threadIdx.x] = a[new_element];
        }
        else if (threadIdx.x == blockDim.x && x < gridDim.x - 1) {
            new_element += 1;
            shm[threadIdx.y + 2][threadIdx.x + 2] = a[new_element];
        }
    }
    if (threadIdx.x == 0 && x > 0) {
        int new_element = tid - 1;
        shm[threadIdx.y + 1][threadIdx.x] = a[new_element];
    }
    if (threadIdx.x == blockDim.x && x < gridDim.x - 1) {
        int new_element = tid + 1;
        shm[threadIdx.y + 1][threadIdx.x + 2] = a[new_element];
    }
    __syncthreads();

    float s = 0;
    int c = 0;
    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j) {
            int new_x = x + dirs[i];
            int new_y = y + dirs[j];
            if (new_x >= 0 && new_x < n && new_y >= 0 && new_y < n) {
                s += shm[threadIdx.y + 1 + dirs[j]][threadIdx.x + 1 + dirs[i]];
                c += 1;
            }
        }
    }
    a[tid] = s / c;
}

__host__ void anti_aliasing(float* a, int n, int k) {
    float* dev_a;
    int* dev_n;
    hipMalloc((void**)&dev_a, n * n * sizeof(float));
    hipMalloc((void**)&dev_n, sizeof(int));
    hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_a, a, n * n * sizeof(float), hipMemcpyHostToDevice);

    int grid_size = n / BLOCK_DIM + 1;
    dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
    dim3 gridDim(grid_size, grid_size);

    for (int i = 0; i < k; ++i) {
        anti_aliasing_gpu<<<gridDim, blockDim>>>(dev_a, dev_n);
    }
    hipMemcpy(a, dev_a, n * n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_n);
}

int main() {
    int n = 6;
    float* slika = (float*)malloc(n * n * sizeof(float));
    for (int i = 0; i < n * n; ++i)
        slika[i] = rand() % 2;


    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%5.2f ", slika[i * n + j]);
        }
        printf("\n");
    }
    printf("\n---------------------------------------------\n");

    anti_aliasing(slika, n, 1);
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%5.2f ", slika[i * n + j]);
        }
        printf("\n");
    }

    free(slika);
}