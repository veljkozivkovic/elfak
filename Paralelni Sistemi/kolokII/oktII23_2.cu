
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256
#define GRID_SIZE 256
int elements_per_reduction_kernel_call = BLOCK_SIZE * GRID_SIZE * 2;

__global__ void get_min_vector(int* a, int* b, int* c, int* n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= *n)
        return;
    
    c[tid] = min(a[tid], b[tid]);
}

__global__ void sum_reduction(int* a, int* b) {
    __shared__ int partial_sum[BLOCK_SIZE];

    int i = threadIdx.x + blockIdx.x * blockDim.x * 2;
    partial_sum[threadIdx.x] = a[i] + a[i + blockDim.x];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        b[blockIdx.x] = partial_sum[0];
}

__host__ void getMatrixC(int** matrixA, int** matrixB, int** matrixC, int* n) {
    int* temp_b = (int*)malloc(*n * sizeof(int));
    int* dev_a, *dev_b, *dev_c, *dev_n;
    hipMalloc((void**)&dev_a, *n * sizeof(int));
    hipMalloc((void**)&dev_b, *n * sizeof(int));
    hipMalloc((void**)&dev_c, *n * sizeof(int));
    hipMalloc((void**)&dev_n, sizeof(int));

    for (int i = 0; i < *n; ++i) {
        for (int j = 0; j < *n; ++j)
            temp_b[j] = matrixB[j][i];
        
        int offset = 0;
        while (offset < *n) {
            int n_of_elements = offset + BLOCK_SIZE * GRID_SIZE <= *n ?
                BLOCK_SIZE * GRID_SIZE : *n - offset;
            hipMemcpy(dev_n, &n_of_elements, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(dev_a, &matrixA[i][offset], n_of_elements * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(dev_b, &temp_b[offset], n_of_elements * sizeof(int), hipMemcpyHostToDevice);

            get_min_vector<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b, dev_c, dev_n);
            hipMemcpy(&matrixC[i][offset], dev_c, n_of_elements * sizeof(int), hipMemcpyDeviceToHost);

            offset += n_of_elements;
        }
    }

    free(temp_b);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_n);
}

__host__ int get_avg(int* a, int n) {
    int* dev_a, *dev_b;
    hipMalloc((void**)&dev_a, elements_per_reduction_kernel_call * sizeof(int));
    hipMalloc((void**)&dev_b, GRID_SIZE * sizeof(int));
    int partial_sum;
    int s = 0;
    for (int i = 0; i < n; i += elements_per_reduction_kernel_call) {
        hipMemcpy(dev_a, &a[i], elements_per_reduction_kernel_call * sizeof(int), hipMemcpyHostToDevice);
        
        sum_reduction<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b);
        sum_reduction<<<1, GRID_SIZE / 2>>>(dev_b, dev_b);

        hipMemcpy(&partial_sum, dev_b, sizeof(int), hipMemcpyDeviceToHost);
        s += partial_sum;
    }

    hipFree(dev_a);
    hipFree(dev_b);
    return s / n;
}

int get_arr_len(int N) {
    if (N < elements_per_reduction_kernel_call)
        return elements_per_reduction_kernel_call;
    
    int p = elements_per_reduction_kernel_call;
    while (p < N)
        p <<= 1;
    
    return p;
}

int main() {
    int n = 555;

    int* a = (int*)malloc(n * n * sizeof(int));
    int* b = (int*)malloc(n * n * sizeof(int));
    int* c = (int*)malloc(n * n * sizeof(int));
    int** matrixA = (int**)malloc(n * sizeof(int*));
    int** matrixB = (int**)malloc(n * sizeof(int*));
    int** matrixC = (int**)malloc(n * sizeof(int*));
    for (int i = 0; i < n; ++i) {
        matrixA[i] = &a[i * n];
        matrixB[i] = &b[i * n];
        matrixC[i] = &c[i * n];
    }

    for(int i = 0; i < n * n; ++i) {
        a[i] = rand() % 20;
        b[i] = rand() % 20;
    }

    getMatrixC(matrixA, matrixB, matrixC, &n);

    // for (int i = 0; i < n * n; ++i) {
    //     printf("%2d ", a[i]);
    // }
    // printf("\n");
    // for (int i = 0; i < n; ++i) {
    //     for (int j = 0; j < n; ++j) {
    //         printf("%2d ", b[j * n + i]);
    //     }
    // }
    // printf("\n");
    // for (int i = 0; i < n * n; ++i) {
    //     printf("%2d ", c[i]);
    // }
    // printf("\n");
    int N = get_arr_len(n);
    int* elements = (int*)malloc(N * sizeof(int));
    int* avg_arr = (int*)malloc(n * sizeof(int));
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j)
            elements[j] = matrixC[j][i];
        for (int j = n; j < N; ++j)
            elements[j] = 0;
        avg_arr[i] = get_avg(elements, n);
    }
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%2d ", matrixC[i][j]);
        }
        printf("\n");
    }
    printf("---------------------------------------------------------------------\n");
    for (int i = 0; i < n; ++i)
        printf("%2d ", avg_arr[i]);
    printf("\n");

    free(a);
    free(b);
    free(c);
    free(matrixA);
    free(matrixB);
    free(matrixC);
    free(elements);
    free(avg_arr);
}