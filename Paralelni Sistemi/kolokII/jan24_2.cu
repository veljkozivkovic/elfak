
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 8

typedef struct Tacka {
    int x;
    int y;
    int index;
} Tacka;

__device__ float distance(Tacka t1, Tacka t2) {
    return sqrtf(powf(t1.x - t2.x, 2) + powf(t1.y - t2.y, 2));
}

__global__ void min_reduction(Tacka* a, Tacka* b, Tacka* ref_tacka) {
    __shared__ Tacka partial_min[BLOCK_SIZE];

    int i = threadIdx.x + blockIdx.x * blockDim.x * 2;

    if (distance(a[i], *ref_tacka) < distance(a[i + blockDim.x], *ref_tacka))
        partial_min[threadIdx.x] = a[i];
    else
        partial_min[threadIdx.x] = a[i + blockDim.x];

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            if (distance(partial_min[threadIdx.x], *ref_tacka) > distance(partial_min[threadIdx.x + s], *ref_tacka))
                partial_min[threadIdx.x] = partial_min[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        b[blockIdx.x] = partial_min[0];
}

__host__ void get_mins(Tacka* a, int n) {
    int grid_size = n / BLOCK_SIZE / 2;

    Tacka* dev_a, *dev_b, *ref_tacka;
    hipMalloc((void**)&dev_a, n * sizeof(Tacka));
    hipMalloc((void**)&dev_b, grid_size * sizeof(Tacka));
    hipMalloc((void**)&ref_tacka, sizeof(Tacka));
    hipMemcpy(ref_tacka, a, sizeof(Tacka), hipMemcpyHostToDevice);
    Tacka min_tacka;

    for (int i = 1; i < 11; ++i) {
        hipMemcpy(dev_a, &a[i], n * sizeof(Tacka), hipMemcpyHostToDevice);
        min_reduction<<<grid_size, BLOCK_SIZE>>>(dev_a, dev_b, ref_tacka);
        min_reduction<<<1, grid_size / 2>>>(dev_b, dev_b, ref_tacka);
        hipMemcpy(&min_tacka, dev_b, sizeof(Tacka), hipMemcpyDeviceToHost);

        // printf("%d %d %d\n", min_tacka.x, min_tacka.y, min_tacka.index);
        a[min_tacka.index] = a[i];
        a[min_tacka.index].index = min_tacka.index;
        a[i] = min_tacka;
        a[i].index = i;
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(ref_tacka);
}

int get_new_size(int n) {
    if (n < 2 * BLOCK_SIZE)
        return 2 * BLOCK_SIZE;
    
    int p = 2 * BLOCK_SIZE;
    while (p < n)
        p <<= 1;
    
    return p;
}

int main() {
    int N = 25;

    int n = get_new_size(N);

    Tacka* tacke = (Tacka*)malloc((n + 12) * sizeof(Tacka));
    for (int i = 0; i < N; ++i) {
        tacke[i].x = i;
        tacke[i].y = i;
        tacke[i].index = i;
    }
    for (int i = N; i < n + 12; ++i) {
        tacke[i].x = sqrt(INT_MAX) - 1;
        tacke[i].y = sqrt(INT_MAX) - 1;
        tacke[i].index = i;
    }
    tacke[0].x = tacke[N - 1].x + 1;
    tacke[0].y = tacke[N - 1].y + 1;

    get_mins(tacke, n);
    for (int i = 1; i < 11; ++i) {
        printf("%d %d\n", tacke[i].x, tacke[i].y);
    }

    free(tacke);
}