#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256
#define GRID_SIZE 256
int elements_per_reduction = BLOCK_SIZE * GRID_SIZE * 2;

__global__ void min_reduction(int* a, int* b) {
    __shared__ int partial_min[BLOCK_SIZE];

    int i = threadIdx.x + blockDim.x * blockIdx.x * 2;
    partial_min[threadIdx.x] = min(a[i], a[i + blockDim.x]);

    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            partial_min[threadIdx.x] = min(partial_min[threadIdx.x], partial_min[threadIdx.x + s]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        b[blockIdx.x] = partial_min[0];
}

__host__ int get_min(int* a, int n) {
    int* dev_a, *dev_b;
    hipMalloc((void**)&dev_a, elements_per_reduction * sizeof(int));
    hipMalloc((void**)&dev_b, GRID_SIZE * sizeof(int));

    int s = INT_MAX;
    int reduced_min;
    for (int i = 0; i < n; i += elements_per_reduction) {
        hipMemcpy(dev_a, &a[i], elements_per_reduction * sizeof(int), hipMemcpyHostToDevice);

        min_reduction<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b);
        min_reduction<<<1, GRID_SIZE / 2>>>(dev_b, dev_b);

        hipMemcpy(&reduced_min, dev_b, sizeof(int), hipMemcpyDeviceToHost);
        s = min(s, reduced_min);
    }

    hipFree(dev_a);
    hipFree(dev_b);

    return s;
}

__host__ int get_new_len(int old_len) {
    if (old_len <= elements_per_reduction)
        return elements_per_reduction;

    int p = elements_per_reduction;
    while (p < old_len)
        p <<= 1;

    return p;
}

int main() {
    int N = 720;
    
    int* matrix = (int*)malloc(N * N * sizeof(int));
    for (int i = 0; i < N * N; ++i)
        matrix[i] = rand();

    int n = get_new_len(N);
    int* a = (int*)malloc(n * sizeof(int));

    int true_min = INT_MAX;
    for (int i = 0; i < N; ++i) {
        a[i] = matrix[i * N + i];
        true_min = min(true_min, a[i]);
    }
    for (int i = N; i < n; ++i) {
        a[i] = INT_MAX;
    }

    int reduced_min = get_min(a, n);

    printf("%d == %d\n", true_min, reduced_min);

    free(a);
    free(matrix);
}