
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256
#define GRID_SIZE 256
int elements_per_reduction = BLOCK_SIZE * GRID_SIZE * 2;

__global__ void get_new_arr(int* a, int* b, int* c, int* n) {
    int tid_a = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid_a < *n * *n) {
        int row = tid_a / *n;
        int col = tid_a % *n;
        int tid_b = col * *n + row;
        
        c[tid_a] = min(a[tid_a], b[tid_b]);
        tid_a += blockDim.x * gridDim.x;
    }    
}

__host__ void get_matrix_c(int* a, int* b, int* c, int* n) {
    int* dev_a, *dev_b, *dev_c, *dev_n;
    hipMalloc((void**)&dev_a, *n * *n * sizeof(int));
    hipMalloc((void**)&dev_b, (*n * *n + 1) * sizeof(int));
    hipMalloc((void**)&dev_c, *n * *n * sizeof(int));
    hipMalloc((void**)&dev_n, sizeof(int));

    hipMemcpy(dev_a, a, *n * *n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, *n * *n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_n, n, sizeof(int), hipMemcpyHostToDevice);

    get_new_arr<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b, dev_c, dev_n);
    hipMemcpy(c, dev_c, *n * *n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_n);
}

__global__ void sum_reduction(int* a, int* b) {
    __shared__ int partial_sum[BLOCK_SIZE];

    int i = threadIdx.x + blockDim.x * blockIdx.x * 2;
    partial_sum[threadIdx.x] = a[i] + a[i + blockDim.x];

    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s)
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        __syncthreads();
    }

    if (threadIdx.x == 0)
        b[blockIdx.x] = partial_sum[0];
}

__host__ void get_col_avgs(int* c, int* arr, int new_n, int old_n) {
    int* dev_a, *dev_b;
    hipMalloc((void**)&dev_a, elements_per_reduction * sizeof(int));
    hipMalloc((void**)&dev_b, GRID_SIZE * sizeof(int));
    int s;

    int* temp = (int*)malloc(new_n * sizeof(int));
    for (int i = old_n; i < new_n; ++i)
        temp[i] = 0;
    
    for (int i = 0; i < old_n; ++i) {
        arr[i] = 0;
        for (int j = 0; j < old_n; ++j) {
            temp[j] = c[j * old_n + i];
        }
        for (int j = 0; j < new_n; j += elements_per_reduction) {
            hipMemcpy(dev_a, &temp[j], elements_per_reduction * sizeof(int), hipMemcpyHostToDevice);

            sum_reduction<<<BLOCK_SIZE, GRID_SIZE>>>(dev_a, dev_b);
            sum_reduction<<<1, GRID_SIZE / 2>>>(dev_b, dev_b);

            hipMemcpy(&s, dev_b, sizeof(int), hipMemcpyDeviceToHost);
            arr[i] += s;
        }
    }

    free(temp);
    hipFree(dev_a);
    hipFree(dev_b);
}

int get_arr_size(int n) {
    if (n <= elements_per_reduction)
        return elements_per_reduction;
    int p = elements_per_reduction;
    while (p < n)
        p <<= 1;
    return p;
}

int main() {
    int N = 5;
    // scanf("%d", &N);
    int* a = (int*)malloc(N * N * sizeof(int));
    int* b = (int*)malloc(N * N * sizeof(int));
    int* c = (int*)malloc(N * N * sizeof(int));

    for (int i = 0; i < N * N; ++i) {
        a[i] = rand() % 50;
        b[i] = rand() % 50;
    }

    get_matrix_c(a, b, c, &N);
    int n = get_arr_size(N);
    int* avg_c = (int*)malloc(n * sizeof(int));
    get_col_avgs(c, avg_c, n, N);

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%4d ", c[i * N + j]);
        }
        printf("\n");
    }
    for (int i = 0; i < N; ++i)
        printf("%4d ", avg_c[i]);
    printf("\n");

    free(a);
    free(b);
    free(c);
    free(avg_c);
}