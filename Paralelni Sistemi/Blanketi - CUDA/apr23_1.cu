
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_DIM 4

__global__ void kernel(int* a, float* b, int* n) {
    __shared__ int shared_mem[BLOCK_DIM + 2][BLOCK_DIM + 2];
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int tid = y * *n + x;
    int dir[3] = {1, 0, -1};

    if (x >= *n || y >= *n)
        return;

    shared_mem[threadIdx.y + 1][threadIdx.x + 1] = a[tid];
    if (threadIdx.y == 0 && y > 0) {
        int new_element = tid - *n;
        shared_mem[threadIdx.y][threadIdx.x + 1] = a[new_element];
        if (threadIdx.x == 0 && x > 0) {
            new_element -= 1;
            shared_mem[threadIdx.y][threadIdx.x] = a[new_element];
        }
        if (threadIdx.x == blockDim.x - 1 && x < gridDim.x - 1) {
            new_element += 1;
            shared_mem[threadIdx.y][threadIdx.x + 2] = a[new_element];
        }
    }
    if (threadIdx.y == blockDim.y - 1 && y < gridDim.y - 1) {
        int new_element = tid + *n;
        shared_mem[threadIdx.y + 2][threadIdx.x + 1] = a[new_element];
        if (threadIdx.x == 0 && x > 0) {
            new_element -= 1;
            shared_mem[threadIdx.y + 2][threadIdx.x] = a[new_element];
        }
        if (threadIdx.x == blockDim.x - 1 && x < gridDim.x - 1) {
            new_element += 1;
            shared_mem[threadIdx.y + 2][threadIdx.x + 2] = a[new_element];
        }
    }
    if (threadIdx.x == 0 && x > 0) {
        int new_element = tid - 1;
        shared_mem[threadIdx.y + 1][threadIdx.x] = a[new_element];
    }
    if (threadIdx.x == blockDim.x - 1 && x < gridDim.x - 1) {
        int new_element = tid + 1;
        shared_mem[threadIdx.y + 1][threadIdx.x + 2] = a[new_element];
    }

    __syncthreads();

    int s = 0;
    int c = 0;
    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j) {
            int new_y = y + dir[i];
            int new_x = x + dir[j];
            if (new_y >= 0 && new_y < *n && new_x >= 0 && new_x < *n) {
                s += shared_mem[threadIdx.y + 1 + dir[i]][threadIdx.x + 1 + dir[j]];
                c += 1;
            }
        }
    }
    b[tid] = (float)s / c;
}

__host__ void initAndCall(int* a, float* b, int n) {
    int* dev_a, *dev_n;
    float* dev_b;
    hipMalloc((void**)&dev_a, n * n * sizeof(int));
    hipMalloc((void**)&dev_b, n * n * sizeof(float));
    hipMalloc((void**)&dev_n, sizeof(int));
    hipMemcpy(dev_a, a, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
    dim3 gridDim(n / BLOCK_DIM + 1, n / BLOCK_DIM + 1);

    kernel<<<gridDim, blockDim>>>(dev_a, dev_b, dev_n);

    hipMemcpy(b, dev_b, n * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_n);
}

int main() {
    int n = 5;
    int* a = (int*)malloc(n * n * sizeof(int));
    float* b = (float*)malloc(n * n * sizeof(float));

    for (int i = 0; i < n * n; ++i)
        a[i] = rand() % 2;

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j)
            printf("%d ", a[i * n + j]);
        printf("\n");
    }
    
    initAndCall(a, b, n);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j)
            printf("%3.2f ", b[i * n + j]);
        printf("\n");
    }

    free(a);
    free(b);
}