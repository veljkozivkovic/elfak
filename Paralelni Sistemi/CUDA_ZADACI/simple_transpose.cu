#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>

#define n 8

__global__ void transpose(int* mat1, int* matT) {
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < n && yIndex < n) {
        int index_in = xIndex + n * yIndex;
        int index_out = yIndex + n * xIndex;
        matT[index_out] = mat1[index_in];
    }

}

int main() {
    int mat[n][n], matT[n][n];
    int* dev_mat, *dev_matT;

    hipMalloc((void**)&dev_mat, n * n * sizeof(int));
    hipMalloc((void**)&dev_matT, n * n * sizeof(int));

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            mat[i][j] = i * n + j;
        }
    }

    hipMemcpy(dev_mat, mat, n * n * sizeof(int), hipMemcpyHostToDevice);

    dim3 gridDef2(2, 2, 1);
    dim3 blockDef2(4, 4, 1);

    transpose<<<gridDef2, blockDef2>>>(dev_mat, dev_matT);
    hipMemcpy(matT, dev_matT, n * n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%d ", matT[i][j]);
        }
        printf("\n");
    }

    hipFree(dev_mat);
    hipFree(dev_matT);
}