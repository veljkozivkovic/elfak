#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>

#define GRID_DIM 32
#define BLOCK_DIM 32

__global__ void kernel(int* a, int* b, int* n) {    
    __shared__ int local_a[BLOCK_DIM + 2];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n[0]) return;
    
    local_a[threadIdx.x] = a[tid];

    if (tid >= n[0] - 2) return; 

    __syncthreads();

    int temp = local_a[threadIdx.x] + local_a[threadIdx.x + 1] + local_a[threadIdx.x + 2];
    b[threadIdx.x] = temp;
}

__host__ void initAndCall(int* a, int* b, int* n) {
    int* dev_a, *dev_b, *dev_n;
    hipMalloc((void**)&dev_a, n[0] * sizeof(int));
    hipMalloc((void**)&dev_b, (n[0] - 2) * sizeof(int));
    hipMalloc((void**)&dev_n, sizeof(int));
    hipMemcpy(dev_a, a, n[0] * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_n, n, sizeof(int), hipMemcpyHostToDevice);

    kernel<<<GRID_DIM, BLOCK_DIM>>>(dev_a, dev_b, dev_n);

    hipMemcpy(b, dev_b, (n[0] - 2) * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_n);
}

int main() {
    int* a, *b, *n;

    n = (int*)malloc(sizeof(int));

    n[0] = 5 + rand() % 15;
    printf("%d:\n", n[0]); 

    a = (int*)malloc(n[0] * sizeof(int));
    b = (int*)malloc((n[0] - 2) * sizeof(int));

    for (int i = 0; i < n[0]; ++i) {
        a[i] = i;
    }

    initAndCall(a, b, n);

    for (int i = 0; i < n[0] - 2; ++i) {
        printf("%d ", b[i]);
    }
    printf("\n");

    free(a);
    free(b);
    free(n);
}