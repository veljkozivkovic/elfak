#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>

#define BLOCK_SIZE 128
#define GRID_SIZE 128

__global__ void kernel(int* A, int* b, int* n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= n[0])
        return;

    int s = 0;
    for (int i = 0; i < n[0]; ++i) {
        s += A[tid + i * n[0]];
    }
    b[tid] = s;
}

__host__ void initAndCall(int* A, int* b, int* n) {
    int* dev_a, *dev_b, *dev_n;
    hipMalloc((void**)&dev_a, n[0] * n[0] * sizeof(int));
    hipMalloc((void**)&dev_b, n[0] * sizeof(int));
    hipMalloc((void**)&dev_n, sizeof(int));
    hipMemcpy(dev_a, A, n[0] * n[0] * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_n, n, sizeof(int), hipMemcpyHostToDevice);

    kernel<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b, dev_n);

    hipMemcpy(b, dev_b, n[0] * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_n);
}

int main() {
    int* n = (int*)malloc(sizeof(int));

    n[0] = 10;
    printf("%d\n", n[0]);

    int* A, *b;
    A = (int*)malloc(n[0] * n[0] * sizeof(int));
    b = (int*)malloc(n[0] * sizeof(int));

    int c = 0;
    for (int i = 0; i < n[0]; ++i) {
        for (int j = 0; j < n[0]; ++j) {
            A[i * n[0] + j] = i * n[0] + j;
        }
    }


    initAndCall(A, b, n);
    for (int i = 0; i < n[0]; ++i) {
        for (int j = 0; j < n[0]; ++j) {
            printf("%4d ", A[i * n[0] + j]);
        }
        printf("\n");
    }

    for (int i = 0; i < n[0]; ++i) {
        printf("%4d ", b[i]);
    }
    printf("\n");

    return 0;
}