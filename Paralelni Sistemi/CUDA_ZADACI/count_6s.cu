#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>

#define n 16

__device__ int isEqualTo6(int num) {
    return num == 6;
}

__global__ void count(int* niz, int* res) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < n) {
        res[tid] = isEqualTo6(niz[tid]);
        tid += blockDim.x + gridDim.x;
    }
}

int main() {
    int niz[n], res_cpu[n];
    int* dev_niz, *res;

    hipMalloc((void**)&dev_niz, n * sizeof(int));
    hipMalloc((void**)&res, n * sizeof(int));

    for (int i = 0; i < n; ++i) {
        niz[i] = rand() % 10;
        printf("%d ", niz[i]);
    }
    printf("\n");

    hipMemcpy(dev_niz, niz, n * sizeof(int), hipMemcpyHostToDevice);

    count<<<4, 4>>>(dev_niz, res);
    hipMemcpy(res_cpu, res, n * sizeof(int), hipMemcpyDeviceToHost);

    int s = 0;
    for (int i = 0; i < n; ++i) {
        printf("%d ", res_cpu[i]);
        s += res_cpu[i];
    }
    printf("\n6's=%d\n", s);
    

    hipFree(dev_niz);
    hipFree(res);
    return 0;
}