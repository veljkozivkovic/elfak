#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>

#define n 8

#define BLOCK_DIM 4
#define GRID_DIM 2

__global__ void transpose(int* mat1, int* matT) {
    __shared__ float block[BLOCK_DIM + 1][BLOCK_DIM + 1];

    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < n && yIndex < n) {
        int index_in = xIndex + n * yIndex;
        block[threadIdx.y][threadIdx.x] = mat1[index_in];
    }

    xIndex = blockIdx.y * blockDim.x + threadIdx.x;
    yIndex = blockIdx.x * blockDim.y + threadIdx.y;
    if (xIndex < n && yIndex < n) {
        int index_out = yIndex * n + xIndex;
        matT[index_out] = block[threadIdx.x][threadIdx.y];
    }

}

int main() {
    int mat[n][n], matT[n][n];
    int* dev_mat, *dev_matT;

    hipMalloc((void**)&dev_mat, n * n * sizeof(int));
    hipMalloc((void**)&dev_matT, n * n * sizeof(int));

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            mat[i][j] = i * n + j;
        }
    }

    hipMemcpy(dev_mat, mat, n * n * sizeof(int), hipMemcpyHostToDevice);

    dim3 gridDef2(2, 2, 1);
    dim3 blockDef2(4, 4, 1);

    transpose<<<gridDef2, blockDef2>>>(dev_mat, dev_matT);
    hipMemcpy(matT, dev_matT, n * n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%d ", matT[i][j]);
        }
        printf("\n");
    }

    hipFree(dev_mat);
    hipFree(dev_matT);
}