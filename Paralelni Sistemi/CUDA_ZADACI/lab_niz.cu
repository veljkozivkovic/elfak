/*
Koristeći CUDA tehnologiju, napisati program koji za dati niz An računa niz Bn-2 na sledeći način:
B[i] = (A[i] + A[i+1] + A[i+2]) / 3
Ilustracija rada programa za n = 5 data je na slici. Veličinu niza A unosi korisnik. Maksimalno redukovati
broj pristupa globalnoj memoriji. Obratiti pažnju na efikasnost paralelizacije. Omogućiti rad programa za
nizove proizvoljne veličine.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 4
#define GRID_SIZE 3

__global__ void kernel(int* a, int* b, int* n) {
    __shared__ int local_a[BLOCK_SIZE + 2];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < *n) {
        local_a[threadIdx.x] = a[tid];

        if (tid >= *n - 2)
            return;
        
        if (threadIdx.x < 2)
            local_a[threadIdx.x + blockDim.x] = a[tid + blockDim.x];

        __syncthreads();

        int temp = (local_a[threadIdx.x] + local_a[threadIdx.x + 1] + local_a[threadIdx.x + 2]) / 3;
        b[tid] = temp;

        tid += BLOCK_SIZE * GRID_SIZE;
    }
}

__host__ void initAndCall(int* a, int* b, int* n) {
    int* dev_a, *dev_b, *dev_n;
    hipMalloc((void**)&dev_a, (*n + 2) * sizeof(int));
    hipMalloc((void**)&dev_b, (*n - 2) * sizeof(int));
    hipMalloc((void**)&dev_n, sizeof(int));

    hipMemcpy(dev_a, a, *n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_n, n, sizeof(int), hipMemcpyHostToDevice);

    kernel<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b, dev_n);

    hipMemcpy(b, dev_b, (*n - 2) * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_n);    
}

int main() {
    int* n = (int*)malloc(sizeof(int));

    *n = 33;
    int* a = (int*)malloc(*n * sizeof(int));
    int* b = (int*)malloc((*n - 2) * sizeof(int));

    for (int i = 0; i < *n; ++i)
        a[i] = 1;
    for (int i = 0; i < *n - 2; ++i)
        b[i] = 0;
   
    initAndCall(a, b, n);

    for (int i = 0; i < *n - 2; ++i)
      printf("%3d ", i + 1);
    printf("\n");
    for (int i = 0; i < *n - 2; ++i)
        printf("%3d ", b[i]);

    free(a);
    free(b);
    free(n);
}