
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 4
#define GRID_SIZE 4
int len_per_reduction = BLOCK_SIZE * GRID_SIZE * 2; 

__global__ void reduction(int* a, int* b) {
    __shared__ int partial_sum[BLOCK_SIZE];

    int i = threadIdx.x + blockDim.x * blockIdx.x * 2;
    partial_sum[threadIdx.x] = a[i] + a[i + blockDim.x];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        b[blockIdx.x] = partial_sum[0];
}

__host__ int sum_array(int* a, int* b, int* n) {
    int* dev_a, *dev_b;
    hipMalloc((void**)&dev_a, len_per_reduction * sizeof(int));
    hipMalloc((void**)&dev_b, len_per_reduction * sizeof(int));

    int s = 0;
    for (int i = 0; i < *n; i += len_per_reduction) {
        hipMemcpy(dev_a, &a[i], len_per_reduction * sizeof(int), hipMemcpyHostToDevice);

        reduction<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b);
        reduction<<<1, GRID_SIZE / 2>>>(dev_b, dev_b);

        hipMemcpy(b, dev_b, len_per_reduction * sizeof(int), hipMemcpyDeviceToHost);
        s += b[0];
    }

    hipFree(dev_a);
    hipFree(dev_b);

    return s;
}

__host__ int new_array_length(int old_len) {
    if (old_len <= len_per_reduction)
        return len_per_reduction;
    
    int p = len_per_reduction;
    while (p < old_len)
        p <<= 1;
    
    return p;
}

int main() {
    int N = 22;
    int n = new_array_length(N);
    int* a = (int*)malloc(n * sizeof(int));
    int* b = (int*)malloc(n * sizeof(int));
    for (int i = 0; i < N; ++i)
        a[i] = 1;
    for (int i = N; i < n; ++i)
        a[i] = 0;
    
    int suma = sum_array(a, b, &n);
    printf("%d\n", suma);

    free(a);
    free(b);
}
