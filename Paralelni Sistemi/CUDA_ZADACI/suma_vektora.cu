#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>

#define n 155

__global__ void add(int* a, int* b, int* c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < n) {
        c[tid] = a[tid] + b[tid];
        tid += gridDim.x * blockDim.x;
    }
}

int main() {
    int a[n], b[n], c[n];
    int* dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, n * sizeof(int));
    hipMalloc((void**)&dev_b, n * sizeof(int));
    hipMalloc((void**)&dev_c, n * sizeof(int));

    for (int i = 0; i < n; ++i) {
        a[i] = i;
        b[i] = i;
    }

    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    add<<<1, 15>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i) {
        printf("%d ", c[i]);
    }
    printf("\n");

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}